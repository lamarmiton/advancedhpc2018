#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4
int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    Timer timerLab1; // We use another timer in lab1, so better to use another one here
    timer.start();
    switch (lwNum) {
        case 1:
	    timerLab1.start();
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timerLab1.getElapsedTimeInMilliSec());
            timerLab1.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
	    printf("labwork 1 OpenMP ellapsed %.1fms\n",lwNum, timerLab1.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    #pragma opm parallele for
    for (int j = 0; j < 100; j++) {             // let's do it 100 times, otherwise it's too fast!
	#pragma opm parallele for
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }

}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
  int nbDevices;	

  printf("Scanning devices ..\n");
  
  hipGetDeviceCount(&nbDevices);	// Get the number of devices


  printf("We got %d devices here\n\n",nbDevices);

  for (int i = 0; i < nbDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device name: %s\n", prop.name);	// Display the name of the device
    printf("Device Number: %d\n", i);	// Display the id of the device
    printf("Number of core : %d\n",getSPcores(prop)); // Display number of core
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount); // Display the number of Multi processor
    printf("Warp Size : %d threads\n", prop.warpSize); // Display the wrapSize
    printf("Memory Clock Rate : %d kHz\n",
           prop.memoryClockRate);	// Display Memory ClockRate
    printf("Memory Bus Width : %d bits\n",
           prop.memoryBusWidth);	// Display Memory bus Width
    printf("Peak Memory Bandwidth : %f GB/s\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1e9);	//Display memory Brandwith
  }  
}

//Making the kernel

__global__ void grayscale(uchar3 *input, uchar3 *output) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x;

}
	
void Labwork::labwork3_GPU() {
	
	// Get the basic variable such as the pixelcount block size etc ..
	int pixelCount = inputImage->width * inputImage->height;
	int blockSize = 1024;
	int numBlock = pixelCount / blockSize;

	uchar3 *devInput;
	uchar3 *devGray;
	
	// Initialize the output image
	outputImage = static_cast<char *>(malloc(pixelCount * 3));
	
	// Allocate the memory in the device for the Deviceinput and the Deviceouput
	hipMalloc(&devInput, pixelCount * sizeof(uchar3));
	hipMalloc(&devGray, pixelCount * sizeof(uchar3));
	
	// Copy from the HostInput to the devInput (here, the image)
	hipMemcpy(devInput, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);
	
	// Do the thing you want to do
	grayscale<<<numBlock, blockSize>>>(devInput, devGray);
	
	// Copy from the DeviceOutput to the HostOutput (here the image in grayscale)
	hipMemcpy(outputImage, devGray,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
	
	// Don't forget to free
	hipFree(devInput);
	hipFree(devGray);
}

__global__ void grayscale2D(uchar3 *input, uchar3 *output, int imageWidth, int imageHeight) {

	// We need to know where we are rigth now
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	// Whecking if we are still in the image
	if(x>=imageWidth) return;
	if(y>=imageHeight) return;
	
	int tid = imageWidth * y + x; // RowSize * y + x

	// We turn the pixel gray
	output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x;

}

void Labwork::labwork4_GPU() {
   	// Get the basic variable such as the pixelcount block size etc ..
	int pixelCount = inputImage->width * inputImage->height;
	dim3 blockSize = dim3(32, 32);
	dim3 gridSize = dim3(ceil(inputImage->width/blockSize.x), ceil(inputImage->height/blockSize.y));

	uchar3 *devInput;
	uchar3 *devGray;
	
	// Initialize the output image
	outputImage = static_cast<char *>(malloc(pixelCount * 3));
	
	// Allocate the memory in the device for the Deviceinput and the Deviceouput
	hipMalloc(&devInput, pixelCount * sizeof(uchar3));
	hipMalloc(&devGray, pixelCount * sizeof(uchar3));
	
	// Copy from the HostInput to the devInput (here, the image)
	hipMemcpy(devInput, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);
	
	// Do the thing you want to do
	grayscale2D<<<gridSize, blockSize>>>(devInput, devGray, inputImage->width, inputImage->height);
	
	// Copy from the DeviceOutput to the HostOutput (here the image in grayscale)
	hipMemcpy(outputImage, devGray,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
	
	// Don't forget to free
	hipFree(devInput);
	hipFree(devGray);
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

void Labwork::labwork5_GPU() {
    
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
